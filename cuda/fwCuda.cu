#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cmath>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <vector>
#include <queue>
#include <limits.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


using namespace std;

#define FW_MAX      50000
#define BLOCK_SIZE  16


int nodes; //number of nodes
int* matrix; //Input matrix
int* FWDistanceMatrix; // Distance matrix for Floyd-Warshall
int* FWPathMatrix; // Path Matrix for Floyd-Warshall




//measuring function
double get_wall_time() {
    struct timeval time;
    if (gettimeofday(&time, NULL)) {
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}
double get_cpu_time() {
    return (double)clock() / CLOCKS_PER_SEC;
}

void TESTPrintMatrix() {
    for (int i = 0; i < nodes; i++) {
        for (int j = 0; j < nodes; j++) {
            cout << FWDistanceMatrix[i * nodes + j] << " ";
        }
        cout << endl;
    }
}

__global__ void GPU_FloydWarshall(int i, int * deviceDistanceMatrix, int * devicePathMatrix, int nodes) {
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idy < nodes && idx < nodes)
    {
        int actuatPosition = idy * nodes + idx;
        int newDistance = deviceDistanceMatrix[idy * nodes + i] + deviceDistanceMatrix[nodes * i + idx];
        int oldDistance = deviceDistanceMatrix[actuatPosition];
        if (newDistance < oldDistance)
        {
            deviceDistanceMatrix[actuatPosition] = newDistance;
            devicePathMatrix[actuatPosition] = devicePathMatrix[i * nodes + idx];
        }
    }
}

void FloydWarshall() {
    int *deviceDistanceMatrix;
    int *devicePathMatrix;

    hipError_t err;
    err = hipSetDevice(0);
    if (err != hipSuccess) cout << "CHYBA!" << endl;

    err = hipMalloc((int**)&deviceDistanceMatrix, nodes * nodes * sizeof(int));
    if (err != hipSuccess) cout << "chyba" << endl;
    err = hipMalloc((int**)&devicePathMatrix, nodes * nodes * sizeof(int));
    if (err != hipSuccess) cout << "chyba" << endl;
    err = hipMemcpy(deviceDistanceMatrix, FWDistanceMatrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) cout << "chyba" << endl;
    err = hipMemcpy(devicePathMatrix, FWPathMatrix, nodes * nodes * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) cout << "chyba" << endl;

    dim3 dimGrid((nodes - 1) / BLOCK_SIZE + 1, (nodes - 1) / BLOCK_SIZE + 1, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

    for ( int i = 0; i < nodes; i++)
    {
        GPU_FloydWarshall <<< dimGrid, dimBlock>>>(i, deviceDistanceMatrix, devicePathMatrix, nodes);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) cout << "Error" << endl;
    }

    hipMemcpy(FWDistanceMatrix, deviceDistanceMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(FWPathMatrix, devicePathMatrix, nodes * nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(devicePathMatrix);
    hipFree(deviceDistanceMatrix);


}

bool loadMatrix(const char * matrixPath) {

    string line;

    ifstream inFile (matrixPath);
    if (!inFile.is_open()) {
        cout << "Wrong path to file" << endl;
        return false;
    }

    getline(inFile, line);
    nodes = atoi(line.c_str());

    FWDistanceMatrix = new int [nodes * nodes];
    FWPathMatrix = new int [nodes * nodes];

    for (int j = 0; j < nodes; j++)
    {
        getline(inFile, line);
        istringstream is(line);
        for (int i = 0; i < nodes; i++)
        {
            is >> FWDistanceMatrix[j * nodes + i];
            if (i != j && FWDistanceMatrix[j * nodes + i] == 0)
                FWDistanceMatrix[j * nodes + i] = FW_MAX;
        }
    }

    //initialize predecessors
    for ( int i = 0; i < nodes; i++)
        for ( int j = 0; j < nodes; j++)
            FWPathMatrix[i * nodes + j] = ( FWDistanceMatrix[i * nodes + j] == FW_MAX ? -1 : i);


    return true;
}



int main( int argc, const char* argv[] )
{

    if ( argc != 2 ) {
        cout << "Bad Input.. 1st parameter: Path to file." << endl;
        return 1;
    }


    loadMatrix(argv[1]);


    //start of measuring
    double wall0 = get_wall_time();
    double cpu0  = get_cpu_time();

    // Initialize CUDA Event
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    FloydWarshall();

    // Finish recording
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elasped time
    hipEventElapsedTime(&elapsedTime, start, stop);
    elapsedTime /= 1000;
    printf ("Timee : %f s\n", elapsedTime);

    //end of measuring
    double wall1 = get_wall_time();
    double cpu1  = get_cpu_time();

    //TESTPrintMatrix();

    //prints results
    /* for (int i = 0; i < ncg.getNodes(); i++) {
         ncg.FWShortestPathFrom(i);
     }*/
    cout << "Wall Time = " << wall1 - wall0 << endl;
    cout << "CPU Time  = " << cpu1  - cpu0  << endl;
    return 0;
}
